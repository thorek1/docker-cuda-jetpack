#include "hip/hip_runtime.h"
#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
  #undef _GLIBCXX_ATOMIC_BUILTINS
  #undef _GLIBCXX_USE_INT128
#endif

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int

#include "main.h"
#include "gpu_common.h"

// Check that dense modules can be properly parsed by nvcc
#include <Eigen/Dense>
#include <iostream>

using namespace Eigen;
using namespace std;

int main1(){
MatrixXf A = MatrixXf::Random(4,4);
MatrixXf B = MatrixXf::Random(4,4);
RealQZ<MatrixXf> qz(A,B); // preallocate space for 4x4 matrices
//qz.compute(A,B);  // A = Q S Z,  B = Q T Z

// print original matrices and result of decomposition
cout << "A:\n" << A << "\n" << "B:\n" << B << "\n";
cout << "S:\n" << qz.matrixS() << "\n" << "T:\n" << qz.matrixT() << "\n";
cout << "Q:\n" << qz.matrixQ() << "\n" << "Z:\n" << qz.matrixZ() << "\n";

// verify precision
cout << "\nErrors:"
  << "\n|A-QSZ|: " << (A-qz.matrixQ()*qz.matrixS()*qz.matrixZ()).norm()
  << ", |B-QTZ|: " << (B-qz.matrixQ()*qz.matrixT()*qz.matrixZ()).norm()
  << "\n|QQ* - I|: " << (qz.matrixQ()*qz.matrixQ().adjoint() - MatrixXf::Identity(4,4)).norm()
  << ", |ZZ* - I|: " << (qz.matrixZ()*qz.matrixZ().adjoint() - MatrixXf::Identity(4,4)).norm()
  << "\n";
return 0;
}

//int main1(){
//ei_test_init_gpu();
//return 0;
//}


//int main(){
//MatrixXf A = MatrixXf::Random(4,4);
//MatrixXf B = MatrixXf::Random(4,4);
//RealQZ<MatrixXf> qz(A,B); // preallocate space for 4x4 matrices


// print original matrices and result of decomposition
//cout << "A:\n" << A << "\n" << "B:\n" << B << "\n";
//cout << "S:\n" << qz.matrixS() << "\n" << "T:\n" << qz.matrixT() << "\n";
//cout << "Q:\n" << qz.matrixQ() << "\n" << "Z:\n" << qz.matrixZ() << "\n";

// verify precision
//cout << "\nErrors:"
//  << "\n|A-QSZ|: " << (A-qz.matrixQ()*qz.matrixS()*qz.matrixZ()).norm()
//  << ", |B-QTZ|: " << (B-qz.matrixQ()*qz.matrixT()*qz.matrixZ()).norm()
//  << "\n|QQ* - I|: " << (qz.matrixQ()*qz.matrixQ().adjoint() - MatrixXf::Identity(4,4)).norm()
//  << ", |ZZ* - I|: " << (qz.matrixZ()*qz.matrixZ().adjoint() - MatrixXf::Identity(4,4)).norm()
//  << "\n";
//return 0;
//}
